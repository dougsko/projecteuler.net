#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <gmp.h>

/*
__device__ int
is_pandigital(char *n)
{
    int i;
    const char *foo = n;
    char *test = "0123456789";
    char *s;
    char c;

    while(*test != '\0')
    {
        c = *test;
        s = c;
        if(strstr(foo, &s) == NULL)
            return 0;
        test++;
    }

    return 1;
}
*/

__device__ int
is_step(char *n)
{
    char *s = n;
    char *p = n;
    char c, d;

    while (*s != '\0'){
        s = p + 1;
        c = *p;
        d = *s;
        //printf("%c - %c = %d\n",c, d, abs(c-d));
        if(abs(c-d) != 1)
            return 0;
        p++;
        s++;
    }
    return 1;
}

__device__ int
get_global_index()
{
    return  blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void
pandigital_step(char **a, char **good, int N)
{
    int index = get_global_index();
    //if(is_pandigital(a[index]) == 1 && is_step(a[index]) == 1)
    if(a[index] == "1")
    {
        good[index] = a[index];
    }
}

int
main(int argc, char **argv)
{
    char **a_h, **a_d, **good_h, **good_d;
    char *max_s = argv[1];
    mpz_t max, i;
    const int N = 10;
    size_t size = N * sizeof(char **);
    int j, k;
    char *string;
    int block_size = 4;
    int n_blocks = N/block_size + (N % block_size == 0 ? 0:1);

    mpz_init(max);
    mpz_init(i);

    mpz_set_ui(i, 0);
    mpz_set_str(max, max_s, 10);
    a_h = (char **)malloc(size);
    good_h = (char **)malloc(size);
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &good_d, size);

    k = 0;
    //while(mpz_cmp(i, max) <= 0)
    //{
        string = mpz_get_str(NULL, 10, i);
        string = argv[1];
        //printf("%s\n",  string);
        a_h[k] = string;
        mpz_add_ui(i, i, 1);
        k++;
    //}

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(good_d, good_h, size, hipMemcpyHostToDevice);

    pandigital_step <<< n_blocks, block_size >>> (a_d, good_d, sizeof(a_d));

    hipMemcpy(good_h, good_d, sizeof(char **)*N, hipMemcpyDeviceToHost);

    j = 0;
    while(good_h[j] != NULL)
    {
        printf("%s\t%s\n", good_h[j], a_h[j]);
        j++;
    }

    free(a_h);
    free(good_h);
    hipFree(good_d);
    hipFree(a_d);
    mpz_clear(max);
    mpz_clear(i);



    return 0;
}
