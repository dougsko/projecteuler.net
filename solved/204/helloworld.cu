// example1.cpp : Defines the entry point for the console application. 
// compile with: nvcc -o helloworld helloworld.cu -lcudart
//
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int
get_global_id()
{
    int index;
    index =  blockIdx.x * blockDim.x + threadIdx.x;
    return index;
}

// Kernel that executes on the CUDA device __global__ void 
__global__ void 
square_array_gpu(float *a, int N) 
{
  int idx = get_global_id();
  int i;
  int count = 0;
  int factors[5];
  int remain;
  for(i = 0; i*i <= a[idx]; i += 2){
	if(remain = fmod(a[idx], i+1) == 0 ){
	  //count++;
	  factors[i] = i;
	}
  }
  for(i = 0; i < sizeof(factors); i++){
	  if(factors[i] > 5){
		  count++;
	  }
  }
  a[idx] = count;
  //a[idx] = a[idx] * a[idx];
}

// main routine that executes on the host int main(void) {
int
main(int argc, char **argv)
{
  float *a_h, *a_d; // Pointer to host & device arrays
  const int N = atoi(argv[1]); // Number of elements in arrays
  size_t size = N * sizeof(float);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  a_h = (float *)malloc(size); // Allocate array on host
  hipMalloc((void **) &a_d, size); // Allocate array on device

  // Initialize host array and copy it to CUDA device
  for (int i=0; i < N; i++) 
  {
      a_h[i] = (float)i;
  }

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

  // Do calculation on device:
  //int block_size = 4;
  //int n_blocks = N/block_size + (N % block_size == 0 ? 0:1);
  square_array_gpu <<< 1, 1000>>> (a_d, sizeof(a_d));

  // Retrieve result from device and store it in host array
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time to generate: %3.3f ms\n", elapsedTime);

  // Print results
  int count = 0;
  for (int i=0; i < N; i++)
  {
	if(a_h[i] == 0){
		count++;
	}
    printf("%d %.0f\n", i, a_h[i]);
  }
  printf("count = %d\n", count);

  // Cleanup
  free(a_h); 
  hipFree(a_d);

  return 0;
}
